#include "hip/hip_runtime.h"
/*This program tries to replicate the code in custom_swe.py
in order to perform SWE equations. */

#include <iostream>
#include <vector>

// --------------- Physical prameters ---------------
#define L_x 1E+6              //Length of domain in x-direction
#define L_y  1E+6             //Length of domain in y-direction
#define g 9.81                //Acceleration of gravity [m/s^2]
#define H  100                //Depth of fluid [m]
#define f_0  1E-4              //Fixed part ofcoriolis parameter [1/s]
#define beta  2E-11            //gradient of coriolis parameter [1/ms]
#define rho_0  1024.0          //Density of fluid [kg/m^3)]
#define tau_0  0.1             //Amplitude of wind stress [kg/ms^2]

// --------------- Computational prameters ---------------
#define N_x 150
#define N_y 150

#define dx L_x/(N_x - 1)                   // Grid spacing in x-direction
#define dy  L_y/(N_y - 1)                   // Grid spacing in y-direction
#define dt  0.1* std::min(dx, dy)/ sqrt(g*H)    // Time step (defined from the CFL condition)
#define anim_interval 20 

//https://stackoverflow.com/questions/27028226/python-linspace-in-c
//Function uses to mimic linspace from NumPy
template<typename T>
std::vector<double> linspace(T start_in, T end_in, int num_in)
{

  std::vector<double> linspaced;

  double start = static_cast<double>(start_in);
  double end = static_cast<double>(end_in);
  double num = static_cast<double>(num_in);

  if (num == 0) { return linspaced; }
  if (num == 1) 
    {
      linspaced.push_back(start);
      return linspaced;
    }

  double delta = (end - start) / (num - 1);

  for(int i=0; i < num-1; ++i)
    {
      linspaced.push_back(start + delta * i);
    }
  linspaced.push_back(end); // I want to ensure that start and end
                            // are exactly the same as the input
  return linspaced;
}

//Function used to mimic transpose from NumPy
std::vector<std::vector<double>> transpose(std::vector<double> input, std::string axis){
  std::vector<std::vector<double>> newVect;

  //transpose on x-axis, implementation is not correct
  if(axis == "x"){

    for(int i = 0; i < input.size(); i++){
      std::vector<double> temp;
      for(int j = 0; j < input.size(); j++)
        temp.push_back(input[i]);
      newVect.push_back(temp);
    }
  }
  //transpose on y-axis, implementation not complete
  else if(axis == "y"){
    for(int i = 0; i < input.size(); i++){

    }

  }

  return newVect;
}


//Simple function used to print one dimentional vectors 
void print_1d_vector(std::vector<double> vec)
{
  std::cout << "[ ";
  for (double d : vec)
    std::cout << d << " ";
  std::cout << "]" << std::endl;
}

//Function used to print two dimentional vectors
void print_2d_vector(std::vector<std::vector<double>> vec){
  
  for(std::vector<double> list: vec){

    std::cout << "[ ";
    for(double d : list)
      std::cout << d << " ";
    std::cout << "]" << std::endl;
  }
}


int main(int argc, char** argv) {

  //Create matrixes that will do calculations 
  std::vector<double> x = linspace(-L_x/2, L_x/2, N_x);
  std::vector<double> y = linspace(-L_y/2, L_y/2, N_y);
  
  //Used to verify if linespace is working correctly
  //std::vector<double> X = {-100.0,-98.65771812,  -97.31543624,  -95.97315436,  -94.63087248,
  //-93.2885906,   -91.94630872,  -90.60402685,  -89.26174497,  -87.91946309,
  //-86.57718121,  -85.23489933,  -83.89261745,  -82.55033557,  -81.20805369,
  //-79.86577181,  -78.52348993,  -77.18120805,  -75.83892617,  -74.4966443,
  //-73.15436242,  -71.81208054,  -70.46979866,  -69.12751678,  -67.7852349,
  //-66.44295302,  -65.10067114,  -63.75838926,  -62.41610738,  -61.0738255,
  //-59.73154362,  -58.38926174,  -57.04697987,  -55.70469799,  -54.36241611,
  //-53.02013423,  -51.67785235,  -50.33557047,  -48.99328859,  -47.65100671,
  //-46.30872483,  -44.96644295,  -43.62416107,  -42.28187919, -40.93959732,
  //-39.59731544,  -38.25503356,  -36.91275168,  -35.5704698,  -34.22818792,
  //-32.88590604,  -31.54362416,  -30.20134228,  -28.8590604,   -27.51677852,
  //-26.17449664,  -24.83221477,  -23.48993289,  -22.14765101,  -20.80536913,
  //-19.46308725,  -18.12080537,  -16.77852349,  -15.43624161,  -14.09395973,
  //-12.75167785,  -11.40939597,  -10.06711409,   -8.72483221,   -7.38255034,
   //-6.04026846,   -4.69798658,   -3.3557047,    -2.01342282,   -0.67114094,
    //0.67114094,    2.01342282,    3.3557047,     4.69798658,    6.04026846,
    //7.38255034,    8.72483221,   10.06711409,   11.40939597,   12.75167785,
   //14.09395973,   15.43624161,   16.77852349,   18.12080537,   19.46308725,
   //20.80536913,   22.14765101,   23.48993289,   24.83221477,   26.17449664,
   //27.51677852,   28.8590604,    30.20134228,   31.54362416,   32.88590604,
   //34.22818792,   35.5704698,    36.91275168,   38.25503356,   39.59731544,
   //40.93959732,   42.28187919,   43.62416107,   44.96644295,   46.30872483,
   //47.65100671,   48.99328859,   50.33557047,   51.67785235,   53.02013423,
   //54.36241611,   55.70469799,   57.04697987,   58.38926174,   59.73154362,
   //61.0738255,    62.41610738,   63.75838926,   65.10067114,   66.44295302,
   //67.7852349,    69.12751678,   70.46979866,   71.81208054,   73.15436242,
   //74.4966443,    75.83892617,   77.18120805,   78.52348993,   79.86577181,
   //81.20805369,   82.55033557,   83.89261745,   85.23489933,   86.57718121,
  //87.91946309,   89.26174497,   90.60402685,   91.94630872,   93.2885906,
  //94.63087248,   95.97315436,   97.31543624,   98.65771812,  100.0 };

  //Do transpose function onto matrixes to accurately represent velocity vectors when plotting
  std::vector<std::vector<double>> newX = transpose(X, "x");
  std::vector<double> Y = X;

  
  //Testing transpose with print statements
  std::cout << dt << std::endl;
  print_1d_vector(X);
  print_1d_vector(newX[0]);


  return 0;
}